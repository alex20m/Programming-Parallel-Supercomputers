#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>


int
main(void)
{
// Check how many MPI tasks we have
  MPI_Init(NULL, NULL);
  int nprocs, pid;
  MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
  MPI_Comm_rank(MPI_COMM_WORLD, &pid);
  if (pid == 0) printf("Num MPI processes: %d\n", nprocs);


// Check how many GPUs we have
  int num_devices;
  hipGetDeviceCount(&num_devices);
  if (pid == 0) printf("Num devices: %d\n", num_devices);

// Where are we assigned to run?
  char processor_name[MPI_MAX_PROCESSOR_NAME];
  int name_len;
  MPI_Get_processor_name(processor_name, &name_len);

// Make the processes to greet the class 
  printf("Hello class from node %s, rank %d out of %d MPI processes\n",
         processor_name, pid, nprocs);

// Make the GPUs greet the class
  
  for (int i = 0; i < num_devices; ++i) {
     hipSetDevice(i);
     printf("Hello class, GPU %d active on node %s\n",i,processor_name);
  }

  MPI_Finalize();

}

